#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void vector_calulate_sales_cu(int *d_sales, int *d_hd, int *d_bt, int *d_sp, int *d_td){
	int i = threadIdx.x;
	d_sales[i] = (d_hd[i] * 30) + (d_bt[i] * 15) + (d_sp[i] * 10) + (d_td[i] * 25);
}

int main(void) {	
	int N = 7;
	int hd[N] = { 3, 2, 0, 3, 4, 10, 8 };
	int bt[N] = { 5, 4, 3, 5, 5, 13, 11 };
	int sp[N] = { 2, 5, 3, 4, 5, 21, 15 };
	int td[N] = { 0, 1, 1, 4, 3, 16, 8 };
	int sales[N];

	int *d_hd, *d_bt, *d_sp, *d_td, *d_sales;
	hipMalloc((void**)&d_hd, sizeof(int)*N);
	hipMalloc((void**)&d_bt, sizeof(int)*N);
	hipMalloc((void**)&d_sp, sizeof(int)*N);
	hipMalloc((void**)&d_td, sizeof(int)*N);
	hipMalloc((void**)&d_sales, sizeof(int)*N);

	hipMemcpy(d_hd, hd, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_bt, bt, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_sp, sp, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_td, td, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_sales, sales, sizeof(int)*N, hipMemcpyHostToDevice);

	vector_calulate_sales_cu<<<1,N>>>(d_sales, d_hd, d_bt, d_sp, d_td);
	
	hipMemcpy(sales, d_sales, sizeof(int) * N, hipMemcpyDeviceToHost);

	hipFree(d_hd);
	hipFree(d_bt);
	hipFree(d_sp);
	hipFree(d_td);

	printf("Sales ");
	for (int i = 0; i < N; i++) {
		printf("%d ", sales[i]);
	}
	printf("\n");	

	return 0;
}
