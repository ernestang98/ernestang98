
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_GPU(int index){
	printf("Hello from GPU%d[%d]!\n", index, threadIdx.x);
}

int main(void) {
	printf("Hello from CPU!\n");

	hello_GPU<<<1,4>>>(1);
	hello_GPU<<<1,6>>>(2);
	hipDeviceSynchronize();

	return 0;
}
