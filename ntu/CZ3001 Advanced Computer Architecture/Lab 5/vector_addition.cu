#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void vector_add_cu(int *d_c, int *d_a, int *d_b, int N){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	d_c[index] = d_a[index] + d_b[index];
}

int main(void) {	
	int N = 4;
	int a[N] = { 22, 13, 16, 5 };
	int b[N] = { 5, 22, 17, 37 };
	int c[N];

	int *d_a, *d_b, *d_c;
	hipMalloc((void**)&d_a, sizeof(int)*N);
	hipMalloc((void**)&d_b, sizeof(int)*N);
	hipMalloc((void**)&d_c, sizeof(int)*N);

	hipMemcpy(d_a, a, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(int)*N, hipMemcpyHostToDevice);

	vector_add_cu<<<1,N>>>(d_c, d_a, d_b, N);
	hipDeviceSynchronize();

	hipMemcpy(c, d_c, sizeof(int) * N, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);

	printf("A ");
	for (int i = 0; i < N; i++) {
		printf("%d ", a[i]);
	}
	printf("\n");

	printf("B ");
	for (int i = 0; i < N; i++) {
		printf("%d ", b[i]);
	}
	printf("\n");	

	printf("C ");
	for (int i = 0; i < N; i++) {
		printf("%d ", c[i]);
	}
	printf("\n");

	return 0;
}
