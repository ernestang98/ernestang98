#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void vector_dot_product_cu(int *d_c, int *d_a, int *d_b){
	__shared__ int tmp[4];
	int i = threadIdx.x;
	tmp[i] = d_a[i] * d_b[i];

	__syncthreads();

	int sum = 0;
	for (int j = 0; j < 4; j++) {
		sum = sum + tmp[j];
	}
	*d_c = sum;
}

int main(void) {	
	int N = 4;
	int a[N] = { 22, 13, 16, 5 };
	int b[N] = { 5, 22, 17, 37 };
	int c;

	int *d_a, *d_b, *d_c;
	hipMalloc((void**)&d_a, sizeof(int)*N);
	hipMalloc((void**)&d_b, sizeof(int)*N);
	hipMalloc((void**)&d_c, sizeof(int));

	hipMemcpy(d_a, a, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(int)*N, hipMemcpyHostToDevice);

	vector_dot_product_cu<<<1,N>>>(d_c, d_a, d_b);
	
	hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);

	printf("A ");
	for (int i = 0; i < N; i++) {
		printf("%d ", a[i]);
	}
	printf("\n");

	printf("B ");
	for (int i = 0; i < N; i++) {
		printf("%d ", b[i]);
	}
	printf("\n");	

	printf("Answer = %d\n", c);

	return 0;
}
